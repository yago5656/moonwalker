#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <stdint.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>

   
#include "Vanity.h"
#include "Base58.h"
#include "Bech32.h"
#include "hash/sha256.h"
#include "hash/sha512.h"
#include "IntGroup.h"
#include "Wildcard.h"
#include "Timer.h"
#include "hash/ripemd160.h"
#include <algorithm>
#include <vector>
#include "SECP256K1.cpp"

#include "GPUGroup.h"
#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUBase58.h"
#include "GPUWildcard.h"
#include "GPUCompute.h"
#include "GPUEngine.h"
  
#define BLOCKS 256
#define THREADS_PER_BLOCK 256
     


__device__ unsigned long long int totThr2 = 0;

__global__ void keyFinderKernel(uint8_t* gTableXCPU, uint8_t* gTableYCPU)
{

//we use atomicadd to verify how many threads are alive, this number is used to define the starting and end ranges for each thread   
atomicAdd(&totThr2, 1);

   //how many threads ?
    __int128_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
	//initial definitions, change your search ranges here. here we are searching for #20
    __int128_t start = 0xD0000;
    __int128_t end =   0xDFFFF;
    __int128_t range =  end - start;
    __int128_t rangeend;
    __int128_t rangestart;

     //calculate the range for each thread	
     rangeend =  (((range / (totThr2 * 1)) * (index + 1)) + start);
     rangestart = (((range / (totThr2 * 1)) * index) + start);
	 
 
//some hashes to search, comment the actual line and uncomment the hash you need to search, don't forget to change start and end ranges above. in this case whe are searching for #20

//     uint8_t aa[20] = { 0x95, 0xa1, 0x56, 0xcd, 0x21, 0xb4, 0xa6, 0x9d, 0xe9, 0x69, 0xeb, 0x67, 0x16, 0x86, 0x4f, 0x4c, 0x8b, 0x82, 0xa8, 0x2a }; //address HASH160 40 bit 
//   uint8_t aa[20] = { 0x68, 0x13, 0x3e, 0x19, 0xb2, 0xdf, 0xb9, 0x03, 0x4e, 0xdf, 0x98, 0x30, 0xa2, 0x00, 0xcf, 0xdf, 0x38, 0xc9, 0x0c, 0xbd }; //address HASH160 61 bit
 //  uint8_t aa[20] = { 0x9a, 0x01, 0x22, 0x60, 0xd0, 0x1c, 0x51, 0x13, 0xdf, 0x66, 0xc8, 0xa8, 0x43, 0x8c, 0x9f, 0x7a, 0x1e, 0x3d, 0x5d, 0xac }; //address HASH160 46 bit
 //  uint8_t aa[20] = { 0x36, 0xaf, 0x65, 0x9e, 0xdb, 0xe9, 0x44, 0x53, 0xf6, 0x34, 0x4e, 0x92, 0x0d, 0x14, 0x3f, 0x17, 0x78, 0x65, 0x3a, 0xe7 }; //address HASH160 52 bit    
//   uint8_t aa[20] = { 0xf0, 0x22, 0x5b, 0xfc, 0x68, 0xa6, 0xe1, 0x7e, 0x87, 0xcd, 0x8b, 0x5e, 0x60, 0xae, 0x3b, 0xe1, 0x8f, 0x12, 0x07, 0x53 }; //address HASH160 45 bit    
//   uint8_t aa[20] = { 0xd1, 0x56, 0x2e, 0xb3, 0x73, 0x57, 0xf9, 0xe6, 0xfc, 0x41, 0xcb, 0x23, 0x59, 0xf4, 0xd3, 0xed, 0xa4, 0x03, 0x23, 0x29 }; //address HASH160 41 bit
//   uint8_t aa[20] = { 0xf6, 0xd6, 0x7d, 0x79, 0x83, 0xbf, 0x70, 0x45, 0x0f, 0x29, 0x5c, 0x9c, 0xb8, 0x28, 0xda, 0xab, 0x26, 0x5f, 0x1b, 0xfa }; //address HASH160 35 bit
//   uint8_t aa[20] = { 0xd3, 0x9c, 0x47, 0x04, 0x66, 0x4e, 0x1d, 0xeb, 0x76, 0xc9, 0x33, 0x1e, 0x63, 0x75, 0x64, 0xc2, 0x57, 0xd6, 0x8a, 0x08 }; //address HASH160 30 bit

    uint8_t aa[20] = { 0xb9, 0x07, 0xc3, 0xa2, 0xa3, 0xb2, 0x77, 0x89, 0xdf, 0xb5, 0x09, 0xb7, 0x30, 0xdd, 0x47, 0x70, 0x3c, 0x27, 0x28, 0x68 }; //address HASH160 20 bit
	 
//   uint8_t aa[20] = { 0x20, 0xd4, 0x5a, 0x6a, 0x76, 0x25, 0x35, 0x70, 0x0c, 0xe9, 0xe0, 0xb2, 0x16, 0xe3, 0x19, 0x94, 0x33, 0x5d, 0xb8, 0xa5 }; //address HASH160 66 bit
//   uint8_t aa[20] = { 0x73, 0x94, 0x37, 0xbb, 0x3d, 0xd6, 0xd1, 0x98, 0x3e, 0x66, 0x62, 0x9c, 0x5f, 0x08, 0xc7, 0x0e, 0x52, 0x76, 0x93, 0x71 }; //address HASH160 67 bit
//   uint8_t aa[20] = { 0xe0, 0xb8, 0xa2, 0xba, 0xee, 0x1b, 0x77, 0xfc, 0x70, 0x34, 0x55, 0xf3, 0x9d, 0x51, 0x47, 0x74, 0x51, 0xfc, 0x8c, 0xfc }; //address HASH160 68 bit 
//   uint8_t aa[20] = { 0x95, 0xa1, 0x56, 0xcd, 0x21, 0xb4, 0xa6, 0x9d, 0xe9, 0x69, 0xeb, 0x67, 0x16, 0x86, 0x4f, 0x4c, 0x8b, 0x82, 0xa8, 0x2a }; //address HASH160 40 bit 
//	 uint8_t aa[20] = { 0x52, 0xe7, 0x63, 0xa7, 0xdd, 0xc1, 0xaa, 0x4f, 0xa8, 0x11, 0x57, 0x8c, 0x49, 0x1c, 0x1b, 0xc7, 0xfd, 0x57, 0x01, 0x37 }; //address HASH160 65 bit 
	
	
	//we will take and compare only the last 8 bytes of hash160
	
    uint64_t hash160Last8Bytesa;
    uint64_t hash160Last8Bytesb;
    uint64_t hash160Last8Bytesb2;
    uint64_t hash160Last8Bytesb3;
 	
    GET_HASH_LAST_8_BYTES(hash160Last8Bytesa, aa);
	
	uint64_t x,y,x1,y1,x2,y2;
	
//the loop, we will test 3 variations of the key (x, x+1, x-1)
while (true) {
    __int128_t ii;
	
    for (ii = rangestart; ii < rangeend; ii++) {

  uint64_t  qx[4]= { 0x000000000000000, 0x000000000000000,0x000000000000000,0x000000000000000 };
  uint64_t  qy[4]= { 0x000000000000000, 0x000000000000000,0x000000000000000,0x000000000000000 };

  uint64_t   qx2[4]= { 0x000000000000000, 0x000000000000000,0x000000000000000,0x000000000000000 };
  uint64_t   qy2[4]= { 0x000000000000000, 0x000000000000000,0x000000000000000,0x000000000000000 };
 
  uint64_t  qx3[4]= { 0x000000000000000, 0x000000000000000,0x000000000000000,0x000000000000000 };
  uint64_t   qy3[4]= { 0x000000000000000, 0x000000000000000,0x000000000000000,0x000000000000000 };

//empty the actual hash160 bytes
    uint64_t hash160Last8Bytesb = 0;
    uint64_t hash160Last8Bytesb2 = 0;
    uint64_t hash160Last8Bytesb3 = 0;
	
//we take the 128 bit integer and split it in two 64 bit numbers to work with uint64 array
y = static_cast<uint64_t>(ii >> 64);
x = static_cast<uint64_t>(ii);

y1 = y;
x1 = x + 1;

y2 = y;
x2 = x - 1;

          //the priv keys
          uint64_t curi1[4] = { x, y, 0x000000000000000, 0x000000000000000 };
          uint64_t curi2[4] = { x1, y1, 0x000000000000000, 0x000000000000000 };
          uint64_t curi3[4] = { x2, y2, 0x000000000000000, 0x000000000000000 };

            //we take the array and turn into uint16 to work with point multiplication
            uint16_t* pv = (uint16_t*)(&curi1);
            uint16_t* pv1 = (uint16_t*)(&curi2);
            uint16_t* pv2 = (uint16_t*)(&curi3);
			
			//point multiplication, we take the integer and multiply by G 
            _PointMultiSecp256k1(qx, qy, pv, gTableXCPU, gTableYCPU);
            _PointMultiSecp256k1(qx2, qy2, pv1, gTableXCPU, gTableYCPU);
            _PointMultiSecp256k1(qx3, qy3, pv2, gTableXCPU, gTableYCPU);

			uint8_t hash160[SIZE_HASH160];
            uint8_t hash1602[SIZE_HASH160];
            uint8_t hash1603[SIZE_HASH160];
			
//is Y odd or even ?
int qy0x = 0;
if (qy[0] % 2) { qy0x = 1; };

int qy1x = 0;
if (qy2[0] % 2) { qy1x = 1; };

int qy2x = 0;
if (qy3[0] % 2) { qy2x = 1; };

            //we take the result and calculate hash160
            _GetHash160Comp(qx, (uint8_t)(qy0x), hash160);
            _GetHash160Comp(qx2, (uint8_t)(qy1x), hash1602);
            _GetHash160Comp(qx3, (uint8_t)(qy2x), hash1603);

            //last 8 bytes
            GET_HASH_LAST_8_BYTES(hash160Last8Bytesb, hash160);
            GET_HASH_LAST_8_BYTES(hash160Last8Bytesb2, hash1602);
            GET_HASH_LAST_8_BYTES(hash160Last8Bytesb3, hash1603);

            //and finally we compare with our hash160, if found the program stops
			
            if (hash160Last8Bytesb == hash160Last8Bytesa) {
                uint64_t xx;
                char foo[20];
                printf("FOUND PRIVKEY 0x%" PRIx64 " 0x%" PRIx64 " 0x % " PRIx64 " \n", (uint64_t)curi1[2], (uint64_t)curi1[1], (uint64_t)curi1[0]);
                asm("trap;");
            }
			
            if (hash160Last8Bytesb2 == hash160Last8Bytesa) {
                uint64_t xx;
                char foo[20];
                printf("FOUND PRIVKEY 0x%" PRIx64 " 0x%" PRIx64 " 0x % " PRIx64 " \n", (uint64_t)curi2[2], (uint64_t)curi2[1], (uint64_t)curi2[0]);
                asm("trap;");
            }

            if (hash160Last8Bytesb3 == hash160Last8Bytesa) {
                uint64_t xx;
                char foo[20];
                printf("FOUND PRIVKEY 0x%" PRIx64 " 0x%" PRIx64 " 0x % " PRIx64 " \n", (uint64_t)curi3[2], (uint64_t)curi3[1], (uint64_t)curi3[0]);
                asm("trap;");
            }


    }
}
            }

#define NUM_GTABLE_CHUNK 16    // Number of GTable chunks that are pre-computed and stored in global memory
#define NUM_GTABLE_VALUE 65536 // Number of GTable values per chunk (all possible states) (2 ^ NUM_GTABLE_CHUNK)
#define SIZE_GTABLE_POINT 32   // Each Point in GTable consists of two 32-byte coordinates (X and Y)
#define COUNT_GTABLE_POINTS (NUM_GTABLE_CHUNK * NUM_GTABLE_VALUE)

void loadGTable(uint8_t* gTableX, uint8_t* gTableY) {
    std::cout << "loadGTable started" << std::endl;
   
    Secp256K1 *secp = new Secp256K1();
    secp->Init2(); 
   
    for (int i = 0; i < NUM_GTABLE_CHUNK; i++)
    {
        for (int j = 0; j < NUM_GTABLE_VALUE - 1; j++)
        {
            int element = (i * NUM_GTABLE_VALUE) + j;
            Point p = secp->GTable2[element];
            for (int b = 0; b < 32; b++) {
                gTableX[(element * SIZE_GTABLE_POINT) + b] = p.x.GetByte64(b);
                gTableY[(element * SIZE_GTABLE_POINT) + b] = p.y.GetByte64(b);
            }
        }
    }

    std::cout << "loadGTable finished!" << std::endl;
}
int main()
{
    printf("MoonWalker YABF v0.2 beta\n");
  hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));

    uint8_t* gTableXCPU = new uint8_t[COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT];
    uint8_t* gTableYCPU = new uint8_t[COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT];
    uint8_t* gTableXGPU;
    uint8_t* gTableYGPU;
    loadGTable(gTableXCPU, gTableYCPU);


   printf("Allocating gTableX \n");
   
    hipMalloc((void**)&gTableXGPU, COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT);
    hipMemset(gTableXGPU, 0, COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT);
    hipMemcpy(gTableXGPU, gTableXCPU, COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT, hipMemcpyHostToDevice);
    printf("Allocating gTableY \n");
    hipMalloc((void**)&gTableYGPU, COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT);
    hipMemset(gTableYGPU, 0, COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT);
    hipMemcpy(gTableYGPU, gTableYCPU, COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT, hipMemcpyHostToDevice);
	
     printf("Go ! \n");
	 
    keyFinderKernel << <BLOCKS, THREADS_PER_BLOCK >> > (gTableXGPU, gTableYGPU);
	
hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();
if (errSync != hipSuccess) 
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

}



